#include "hip/hip_runtime.h"
#ifdef _WIN32
#  define NOMINMAX 
#endif

#define NUM_ELEMENTS 1000000

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <iostream>
// includes, kernels
#include "vector_reduction_kernel.cu"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

int ReadFile(unsigned int*, char* file_name);
unsigned int computeOnDevice(unsigned int* h_data, int array_mem_size);

extern "C" 
void computeGold( unsigned int* reference, unsigned int* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int 
main( int argc, char** argv) 
{
    runTest( argc, argv);
    return EXIT_SUCCESS;
}

////////////////////////////////////////////////////////////////////////////////
//! Run naive scan test
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    int num_elements = NUM_ELEMENTS;
    int errorM = 0;

    const unsigned int array_mem_size = sizeof( unsigned int) * num_elements;

    // allocate host memory to store the input data
    unsigned int* h_data = (unsigned int*) malloc( array_mem_size);

    // * No arguments: Randomly generate input data and compare against the 
    //   host's result.
    // * One argument: Read the input data array from the given file.
    switch(argc-1)
    {      
        case 1:  // One Argument
            errorM = ReadFile(h_data, argv[1]);
            if(errorM != num_elements)
            {
                printf("Error reading input file!\n");
                exit(1);
            }
        break;
        
        default:  // No Arguments or one argument
            // initialize the input data on the host to be integer values
            // between 0 and 1000
            for( unsigned int i = 0; i < num_elements; ++i) 
            {
                //h_data[i] = floorf(1000*(rand()/(float)RAND_MAX));
		h_data[i] = rand()%1000;
            }
        break;  
    }
    // compute reference solution
    unsigned int reference = 0;  
    computeGold(&reference , h_data, num_elements);
    
    // **===-------- Modify the body of this function -----------===**
    unsigned int result = computeOnDevice(h_data, num_elements);
    // **===-----------------------------------------------------------===**


    // We can use an epsilon of 0 since values are integral and in a range 
    // that can be exactly represented
    unsigned int epsilon = 0;
    unsigned int result_regtest = (abs(result - reference) <= epsilon);
    printf( "Test %s\n", (1 == result_regtest) ? "PASSED" : "FAILED");
    //printf( "device: %f  host: %f\n", result, reference);
    printf( "device: %d  host: %d\n", result, reference);
    // cleanup memory
    free( h_data);
}

// Read a vector into M (already allocated) from file
int ReadFile(unsigned int* V, char* file_name)
{
    unsigned int data_read = NUM_ELEMENTS;
    FILE* input = fopen(file_name, "r");
    unsigned i = 0;
    for (i = 0; i < data_read; i++) 
        fscanf(input, "%d", &(V[i]));
    return data_read;
}

// **===----------------- Modify this function ---------------------===**
// Take h_data from host, copies it to device, setup grid and thread 
// dimentions, excutes kernel function, and copy result of reduction back
// to h_data.
// Note: unsigned int* h_data is both the input and the output of this function.
unsigned int computeOnDevice(unsigned int* h_data, int num_elements)
{
    unsigned int* h_device;
    hipMalloc((void**)&h_device, num_elements*sizeof(unsigned int));
    hipMemcpy(h_device, h_data, num_elements*sizeof(unsigned int), hipMemcpyHostToDevice);
    
    //Initializing block sizes
    dim3 grid, block;
    block.x = block_size;
    block.y = 1;
    block.z = 1;
    if(num_elements%(2*block.x) == 0)
        grid.x = num_elements/(2*block.x);
    else
	grid.x = num_elements/(2*block.x) + 1;

    std::cout<<"No of blocks "<<grid.x<<std::endl;

    reduction<<<grid,block>>>(h_device, num_elements);

    hipMemcpy(h_data, h_device, num_elements*sizeof(unsigned int), hipMemcpyDeviceToHost);

    for(int i = 1; i < grid.x; i++)
    {
	h_data[0] += h_data[i];
    }

  // placeholder
  return h_data[0];
  // placeholder
}
